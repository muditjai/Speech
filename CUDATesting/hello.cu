
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void helloCalledFromDevice()
{
	printf("Device fn hello from GPU\n");
}

__global__ void helloFromGPU()
{
	printf("Hello from GPU thread %d\n", threadIdx.x);
	//helloCalledFromDevice();
}
int main()
{
	printf("Hello from CPU\n");
	helloFromGPU<<<2, 5>>>();
	hipDeviceSynchronize();
}
